#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define NUM_THREADS 512
//https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void bitonic_sort_default(int *arr, unsigned int log_len) 
{
	int size = 1 << log_len;

	for (int seq = 2; seq <= size; seq <<= 2) 
	{
		for (int dist = seq >> 2; dist > 0; dist >>= 2) 
		{
			for (int item = 0; item < size; item++) 
			{
				int pair_item = item | dist;

				if (((item & seq) == 0) && (arr[item] > arr[pair_item])
					|| ((item & seq) != 0) && (arr[item] < arr[pair_item]))
				{
					int temp = arr[item];
					arr[item] = arr[pair_item];
					arr[pair_item] = temp;
				}
			}
		}
	}
}

__global__ void bitonic_sort_step(int *arr, int dist, int seq) 
{
	int item = threadIdx.x + blockIdx.x * blockDim.x;
	int pair_item = item | dist;

	if (((item & seq) == 0) && (arr[item] > arr[pair_item])
		|| ((item & seq) != 0) && (arr[item] < arr[pair_item]))
	{
		int temp = arr[item];
		arr[item] = arr[pair_item];
		arr[pair_item] = temp;
	}
}

void bitonic_sort_gpu(int *arr, unsigned int exp) 
{
	int arr_len = 1 << exp;
	size_t arr_size = sizeof(int) * arr_len;
	int* d_arr;

	gpuErrCheck(hipMalloc(&d_arr, arr_size));
	gpuErrCheck(hipMemcpy(d_arr, arr, arr_size, hipMemcpyHostToDevice));

	int num_blocks = arr_len / NUM_THREADS;
	int num_threads = NUM_THREADS;

	if (arr_len / NUM_THREADS == 0) 
	{
		num_blocks = arr_len;
		num_threads = 1;
	}

	for (int seq = 2; seq <= arr_len; seq <<= 1)
	{
		for (int dist = seq >> 1; dist > 0; dist >>= 1) 
		{
			bitonic_sort_step<<<num_blocks, num_threads>>>(d_arr, dist, seq);
		}
	}

	gpuErrCheck(hipMemcpy(arr, d_arr, arr_size, hipMemcpyDeviceToHost));
	hipFree(d_arr);
}
