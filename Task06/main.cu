#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "bitonic_sorts.cuh"

#define TIME_TESTS 5
#define MAX_RAND 1000000
#define MAX_EXP 25

void generate_arr(int *arr, int n)
{
	srand(time(NULL));
	for (int i = 0; i < n; i++)
		arr[i] = (rand() % MAX_RAND);
}

void verify(int *arr, int n)
{
	for (int i = 0; i < n - 1; i++)
		if (arr[i] > arr[i + 1])
		{
			printf("ERROR IN SORT!!!1\n");
			return;
		}

	printf("Sort is correct.\n");
}

int main()
{
	for (unsigned int exp = 1; exp <= MAX_EXP; exp++)
	{
		int len = 1 << exp;
		int* test_arr = (int *) malloc(sizeof(int) * len);	

		if (test_arr == NULL)
		{
			printf("Cannot allocate memory for test array.\n");
			exit(1);
		}
		
		generate_arr(test_arr, len);

		clock_t start, end;
		double cpu_time_used = 0;
		double gpu_time_used = 0;

		for (int i = 0; i < TIME_TESTS; i++)
		{
			int* arr_cpu_copy = (int *) malloc(sizeof(int) * len);
			int* arr_gpu_copy = (int *) malloc(sizeof(int) * len);

			memcpy(arr_cpu_copy, test_arr, len);
			memcpy(arr_gpu_copy, test_arr, len);

			start = clock();
			bitonic_sort_default(arr_cpu_copy, exp);
			end = clock();

			cpu_time_used += ((double)(end - start)) / CLOCKS_PER_SEC;

			start = clock();
			bitonic_sort_gpu(arr_gpu_copy, exp);
			end = clock();

			gpu_time_used += ((double)(end - start)) / CLOCKS_PER_SEC;

			free(arr_cpu_copy);
			free(arr_gpu_copy);
		}

		double cpu_time_avg = cpu_time_used / TIME_TESTS;
		double gpu_time_avg = gpu_time_used / TIME_TESTS;

		printf("%d %f %f\n", len, cpu_time_avg, gpu_time_avg);		

		bitonic_sort_gpu(test_arr, exp);
		verify(test_arr, len);

		free(test_arr);
	}

	return 0;
}